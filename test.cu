#include <hip/hip_runtime.h>
#include "cuda_check.h"

int main(void) {
  // linear gpu0_buffer layout
  constexpr size_t size = 1024;

  char host_buffer[size];
  memset(host_buffer, 0x00, size);

  // select the first device
  CUDA_CHECK(hipSetDevice(0));

  // create a CUDA stream on the first device
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  // ===========================================================================

  // allocate device memory on the first device
  char* gpu0_buffer = nullptr;
  CUDA_CHECK(hipMalloc(&gpu0_buffer, size));

  // select the second device
  CUDA_CHECK(hipSetDevice(1));

  // allocate device memory on the second device
  char* gpu1_buffer = nullptr;
  CUDA_CHECK(hipMalloc(&gpu1_buffer, size));

  // hipMemcpy variants can be called with memory a different device
  CUDA_CHECK(hipMemcpy(host_buffer, gpu0_buffer, size, hipMemcpyDefault));
  CUDA_CHECK(hipMemcpy(gpu0_buffer, host_buffer, size, hipMemcpyDefault));

  CUDA_CHECK(hipMemcpy(host_buffer, gpu1_buffer, size, hipMemcpyDefault));
  CUDA_CHECK(hipMemcpy(gpu1_buffer, host_buffer, size, hipMemcpyDefault));

  CUDA_CHECK(hipMemcpy(gpu1_buffer, gpu0_buffer, size, hipMemcpyDefault));
  CUDA_CHECK(hipMemcpy(gpu0_buffer, gpu1_buffer, size, hipMemcpyDefault));

  // hipMemcpyAsync variants can be called with memory a different device
  CUDA_CHECK(hipMemcpyAsync(host_buffer, gpu0_buffer, size, hipMemcpyDefault, stream));
  CUDA_CHECK(hipMemcpyAsync(gpu0_buffer, host_buffer, size, hipMemcpyDefault, stream));

  CUDA_CHECK(hipMemcpyAsync(host_buffer, gpu1_buffer, size, hipMemcpyDefault, stream));
  CUDA_CHECK(hipMemcpyAsync(gpu1_buffer, host_buffer, size, hipMemcpyDefault, stream));

  CUDA_CHECK(hipMemcpyAsync(gpu1_buffer, gpu0_buffer, size, hipMemcpyDefault, stream));
  CUDA_CHECK(hipMemcpyAsync(gpu0_buffer, gpu1_buffer, size, hipMemcpyDefault, stream));

  // free the device memory
  CUDA_CHECK(hipFree(gpu0_buffer));

  // ===========================================================================

  // select the first device
  CUDA_CHECK(hipSetDevice(0));

  // allocate stream-ordered device memory on the first device
  CUDA_CHECK(hipMallocAsync(&gpu0_buffer, size, stream));

  // select the second device
  CUDA_CHECK(hipSetDevice(1));

  // hipMemcpy variants **cannot** be called with stream-ordered memory on a different device
  CUDA_CHECK(hipMemcpy(host_buffer, gpu0_buffer, size, hipMemcpyDefault));
  CUDA_CHECK(hipMemcpy(gpu0_buffer, host_buffer, size, hipMemcpyDefault));

  CUDA_CHECK(hipMemcpy(host_buffer, gpu1_buffer, size, hipMemcpyDefault));
  CUDA_CHECK(hipMemcpy(gpu1_buffer, host_buffer, size, hipMemcpyDefault));

  // stream-ordered memory **cannot** be copied across devices
  CUDA_CHECK(hipMemcpy(gpu1_buffer, gpu0_buffer, size, hipMemcpyDefault));
  CUDA_CHECK(hipMemcpy(gpu0_buffer, gpu1_buffer, size, hipMemcpyDefault));

  // hipMemcpyAsync variants **cannot** be called with stream-ordered memory on a different device
  CUDA_CHECK(hipMemcpyAsync(host_buffer, gpu0_buffer, size, hipMemcpyDefault, stream));
  CUDA_CHECK(hipMemcpyAsync(gpu0_buffer, host_buffer, size, hipMemcpyDefault, stream));

  CUDA_CHECK(hipMemcpyAsync(host_buffer, gpu1_buffer, size, hipMemcpyDefault, stream));
  CUDA_CHECK(hipMemcpyAsync(gpu1_buffer, host_buffer, size, hipMemcpyDefault, stream));

  // stream-ordered memory **cannot** be copied across devices
  CUDA_CHECK(hipMemcpyAsync(gpu1_buffer, gpu0_buffer, size, hipMemcpyDefault, stream));
  CUDA_CHECK(hipMemcpyAsync(gpu0_buffer, gpu1_buffer, size, hipMemcpyDefault, stream));

  // free the stream-ordered device memory
  CUDA_CHECK(hipFreeAsync(gpu0_buffer, stream));

  // ===========================================================================

  CUDA_CHECK(hipStreamDestroy(stream));
}
