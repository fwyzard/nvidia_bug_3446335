#include <hip/hip_runtime.h>
#include "cuda_check.h"

int main(void) {
  // linear gpu0_buffer layout
  constexpr size_t size = 1024;

  char host_buffer[size];
  memset(host_buffer, 0x00, size);

  // create a CUDA stream on the first device
  hipStream_t stream0;
  CUDA_CHECK(hipSetDevice(0));
  CUDA_CHECK(hipStreamCreate(&stream0));

  // create a CUDA stream on the second device
  hipStream_t stream1;
  CUDA_CHECK(hipSetDevice(1));
  CUDA_CHECK(hipStreamCreate(&stream1));

  // ===========================================================================

  // allocate device memory on the first device
  char* gpu0_buffer = nullptr;
  CUDA_CHECK(hipSetDevice(0));
  CUDA_CHECK(hipMalloc(&gpu0_buffer, size));

  // allocate device memory on the second device
  char* gpu1_buffer = nullptr;
  CUDA_CHECK(hipSetDevice(1));
  CUDA_CHECK(hipMalloc(&gpu1_buffer, size));

  // hipMemcpy variants can be called with memory a different device
  CUDA_CHECK(hipMemcpy(host_buffer, gpu0_buffer, size, hipMemcpyDefault));
  CUDA_CHECK(hipMemcpy(gpu0_buffer, host_buffer, size, hipMemcpyDefault));

  CUDA_CHECK(hipMemcpy(host_buffer, gpu1_buffer, size, hipMemcpyDefault));
  CUDA_CHECK(hipMemcpy(gpu1_buffer, host_buffer, size, hipMemcpyDefault));

  CUDA_CHECK(hipMemcpy(gpu1_buffer, gpu0_buffer, size, hipMemcpyDefault));
  CUDA_CHECK(hipMemcpy(gpu0_buffer, gpu1_buffer, size, hipMemcpyDefault));

  // hipMemcpyAsync variants can be called with memory a different device
  CUDA_CHECK(hipMemcpyAsync(host_buffer, gpu0_buffer, size, hipMemcpyDefault, stream0));
  CUDA_CHECK(hipMemcpyAsync(gpu0_buffer, host_buffer, size, hipMemcpyDefault, stream0));

  CUDA_CHECK(hipMemcpyAsync(host_buffer, gpu1_buffer, size, hipMemcpyDefault, stream0));
  CUDA_CHECK(hipMemcpyAsync(gpu1_buffer, host_buffer, size, hipMemcpyDefault, stream0));

  CUDA_CHECK(hipMemcpyAsync(gpu1_buffer, gpu0_buffer, size, hipMemcpyDefault, stream0));
  CUDA_CHECK(hipMemcpyAsync(gpu0_buffer, gpu1_buffer, size, hipMemcpyDefault, stream0));

  // device memory can be freed with a different current device
  CUDA_CHECK(hipFree(gpu0_buffer));
  CUDA_CHECK(hipFree(gpu1_buffer));

  // ===========================================================================

  // allocate stream-ordered device memory on the first device (determined by the stream)
  CUDA_CHECK(hipMallocAsync(&gpu0_buffer, size, stream0));

  // allocate stream-ordered device memory on the second device (determined by the stream)
  CUDA_CHECK(hipMallocAsync(&gpu1_buffer, size, stream1));

  // hipMemcpy variants **cannot** be called with stream-ordered memory on a different device,
  // select the correct device first
  CUDA_CHECK(hipSetDevice(0));
  CUDA_CHECK(hipMemcpy(host_buffer, gpu0_buffer, size, hipMemcpyDefault));
  CUDA_CHECK(hipMemcpy(gpu0_buffer, host_buffer, size, hipMemcpyDefault));

  CUDA_CHECK(hipSetDevice(1));
  CUDA_CHECK(hipMemcpy(host_buffer, gpu1_buffer, size, hipMemcpyDefault));
  CUDA_CHECK(hipMemcpy(gpu1_buffer, host_buffer, size, hipMemcpyDefault));

  // stream-ordered memory **cannot** be copied across devices with hipMemcpy,
  // use hipMemcpyPeer
  CUDA_CHECK(hipMemcpyPeer(gpu1_buffer, 1, gpu0_buffer, 0, size));
  CUDA_CHECK(hipMemcpyPeer(gpu0_buffer, 0, gpu1_buffer, 1, size));

  // hipMemcpyAsync variants **cannot** be called with stream-ordered memory on a different device,
  // select the correct device first
  CUDA_CHECK(hipSetDevice(0));
  CUDA_CHECK(hipMemcpyAsync(host_buffer, gpu0_buffer, size, hipMemcpyDefault, stream0));
  CUDA_CHECK(hipMemcpyAsync(gpu0_buffer, host_buffer, size, hipMemcpyDefault, stream0));

  CUDA_CHECK(hipSetDevice(1));
  CUDA_CHECK(hipMemcpyAsync(host_buffer, gpu1_buffer, size, hipMemcpyDefault, stream0));
  CUDA_CHECK(hipMemcpyAsync(gpu1_buffer, host_buffer, size, hipMemcpyDefault, stream0));

  // stream-ordered memory **cannot** be copied across devices with hipMemcpyAsync
  // use hipMemcpyPeerAsync
  CUDA_CHECK(hipMemcpyPeerAsync(gpu1_buffer, 1, gpu0_buffer, 0, size, stream0));
  CUDA_CHECK(hipMemcpyPeerAsync(gpu0_buffer, 0, gpu1_buffer, 1, size, stream0));

  // stream-ordered device memory can be freed with hipFree (without further synchronisation),
  // or with hipFreeAsync (passing the stream where the operation should be ordered, which can be
  // for a different device than the memory pool)
  CUDA_CHECK(hipFreeAsync(gpu0_buffer, stream0));
  CUDA_CHECK(hipFreeAsync(gpu1_buffer, stream0));

  // ===========================================================================

  CUDA_CHECK(hipStreamDestroy(stream0));
  CUDA_CHECK(hipStreamDestroy(stream1));
}
